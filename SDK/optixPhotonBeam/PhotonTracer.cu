#include "hip/hip_runtime.h"
#include <optix_device.h>
#include <sutil/vec_math.h>
#include "random.h"
#include "LaunchParams.h"

extern "C" __constant__ PhotonBeamParams optixLaunchParams;

//------------------------------------------------------------------------------
// closest hit and anyhit programs for radiance-type rays.
//
// Note eventually we will have to create one pair of those for each
// ray type and each geometry type we want to render; but this
// simple example doesn't use any actual geometries yet, so we only
// create a single, dummy, set of them (we do have to have at least
// one group of them to set up the SBT)
//------------------------------------------------------------------------------

extern "C" __global__ void __closesthit__radiance()
{ /*! for this simple example, this will remain empty */
}

extern "C" __global__ void __anyhit__radiance()
{ /*! for this simple example, this will remain empty */
}



//------------------------------------------------------------------------------
// miss program that gets called for any ray that did not have a
// valid intersection
//
// as with the anyhit/closest hit programs, in this example we only
// need to have _some_ dummy function to set up a valid SBT
// ------------------------------------------------------------------------------

extern "C" __global__ void __miss__radiance()
{ /*! for this simple example, this will remain empty */
}

//------------------------------------------------------------------------------
// ray gen program - the actual rendering happens in here
//------------------------------------------------------------------------------
extern "C" __global__ void __raygen__renderFrame()
{
    if (optixGetLaunchIndex().x == 0 &&
        optixLaunchParams.seed % 1000 == 0) {
        // we could of course also have used optixGetLaunchDims to query
        // the launch size, but accessing the optixLaunchParams here
        // makes sure they're not getting optimized away (because
        // otherwise they'd not get used)
        printf("############################################\n");
        printf("Hello world from OptiX 7 raygen program!\n(within a %ix%i-sized launch)\n",
            optixLaunchParams.maxBeams, optixLaunchParams.seed);
        printf("############################################\n");
    }
    
    // ------------------------------------------------------------------
    // for this example, produce a simple test pattern:
    // ------------------------------------------------------------------
    
    // compute a test pattern based on pixel ID
    const uint3 idx = optixGetLaunchIndex();
    int launchSeed = optixLaunchParams.seed;
    unsigned int seed = tea<4>(idx.x, launchSeed);
    //float3 start = make_float3(rnd(seed) * 2.f - 1.f, rnd(seed) * 2.f - 1.f, rnd(seed) * 2.f - 1.f) / 10.f;
    float3 start = make_float3(0.f, 0.f, 0.f);
    float transmittance = 2.f;
    float thickness = 0.01f;
    float mult = 1.f;
    float alpha = 0.1f;
    for (int j = 0; j < launchSeed; j++)
        mult = mult * (j + alpha + 1) / (j + 1);
    mult = mult / (launchSeed + 1);
    thickness = thickness * mult;
    for (int i = 0; i < optixLaunchParams.maxBounce; i++) {
        optixLaunchParams.beams[idx.x * optixLaunchParams.maxBounce + i].transmittance = transmittance;
        optixLaunchParams.beams[idx.x * optixLaunchParams.maxBounce + i].start = start;
        float u = 2.f * rnd(seed) - 1.f;
        float v = 2.f * rnd(seed) - 1.f;
        float theta = M_PI / 2.f + M_PI * pow(u, 3);
        float phi = (M_PI + M_PI * pow(v, 3)) / 2.f;
        float3 dir = make_float3(cos(theta) * sin(phi), sin(theta) * sin(phi), cos(phi));
        float eta = rnd(seed);
        float t = (-1.0f * log(1 - eta)) / optixLaunchParams.materialProp;
        float3 end = start + t * dir;
        optixLaunchParams.beams[idx.x * optixLaunchParams.maxBounce + i].end = start + t * dir;
        optixLaunchParams.beams[idx.x * optixLaunchParams.maxBounce + i].thickness = thickness;
        start = end;
        transmittance = transmittance * exp(-t * optixLaunchParams.materialProp);
    }
    // and write to frame buffer ...
}